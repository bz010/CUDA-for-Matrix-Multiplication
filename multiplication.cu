#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// DO NOT change the kernel function
__global__ void vector_add(int *a, int *b, int *c)
{
// DO NOT change the kernel function
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}


#define N (2048*2048)
#define THREADS_PER_BLOCK 128

int main()
{
	int* a, * b, * c, * golden;
	int* d_a, * d_b, * d_c;
	int size = N * sizeof(int);

	// from class
	#define NSTREAMS 4
	int nsdata = N / NSTREAMS;
	int iBytes = nsdata * sizeof(float);
    // ngrid.x = (nsdata + nblock.x - 1) / nblock.x;
	hipStream_t streams[NSTREAMS];
	for (int i = 0; i < NSTREAMS; i++)
	{
		hipStreamCreate(&streams[i]);
	}
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	golden = (int *)malloc(size);

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		golden[i] = a[i] + b[i];
		c[i] = 0;
	}
    // same for loop from lecture with some sweaks in how data is read since pointer was given
	// Async dma no time event(waitevent)
	for (int i = 0; i < NSTREAMS; i++)
	{
		int offset = i * nsdata;
		hipMemcpyAsync(d_a + offset, a + offset, iBytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(d_b + offset, b + offset, iBytes, hipMemcpyHostToDevice, streams[i]);
		vector_add <<<(nsdata + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK, 0, streams[i] >>> (d_a + offset, d_b + offset, d_c + offset);
		hipMemcpyAsync(c + offset, d_c + offset, iBytes, hipMemcpyDeviceToHost, streams[i]);
	}

	hipDeviceSynchronize();
	

	bool pass = true;
	for (int i = 0; i < N; i++) {
		if (golden[i] != c[i])
			pass = false;
	}
	
	if (pass)
		printf("PASS\n");
	else
		printf("FAIL\n");

	printf("Ben Zhang, A16268103\n");

	free(a);
	free(b);
	free(c);
	free(golden);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 
